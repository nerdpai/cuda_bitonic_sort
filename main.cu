
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <time.h>
#include <math.h>
#include <limits>

struct RandomSpecs
{
  static constexpr int multiplier = 8253729;
  static constexpr int addend = 2396403;
  static constexpr int mod = 32767;
};


__host__ __device__
int random_int(int seed) {
  seed = (RandomSpecs::multiplier * seed + RandomSpecs::addend);
  return abs(seed % RandomSpecs::mod);
}

int log2_ceil(int n){
  float power2 = log2(n);
  float floor_power2 = floor(power2);
  if (abs(power2 - floor_power2) > std::numeric_limits<float>::epsilon()) {
    floor_power2 += 1;
  }

  return floor_power2;
}

__global__
void assign_random(int* x, int seed)
{
  int index = threadIdx.x;
  int rand_num = random_int(seed + index);
  x[index] = rand_num;
}

__global__
void assign_constant_from(int* x, int constant, int from)
{
  int index = threadIdx.x;
  x[index+from] = constant;
}

__global__
void bitonic_stage(int* x, int step, int stage)
{
  int thread_idx = threadIdx.x;
  int distance = int(pow(2, stage-1));
  int repeativness = int(pow(2, step-1));
  int first_idx = thread_idx + (thread_idx / distance) * distance;
  int second_idx = first_idx + distance;

  int first, second;
  first = min(x[first_idx], x[second_idx]);
  second = max(x[first_idx], x[second_idx]);
  int mod2 = (thread_idx / repeativness) % 2;
  if (mod2 % 2 == 1){
    int temp = first;
    first = second;
    second = temp;
  }

  x[first_idx] = first;
  x[second_idx] = second;
}

void print_array(int* x, int N)
{
  for (int i = 0; i < N; i++){
      std::cout << x[i] << ", ";
  }
  std::cout << std::endl;
}

void bitonic_sort(int* x, int N) // N should be a power of 2
{
  int power2 = int(log2(N));
  int workers = N/2;
  for (int step = 1; step <= power2; step++){
    for (int stage = step; stage > 0; stage--){
      bitonic_stage<<<1, workers>>>(x, step, stage);
      hipDeviceSynchronize();
    }
  }
}

int main(void)
{
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int max_threads_num = prop.maxThreadsPerBlock;
  while(true){
    int N = 0;
    int real_size = 0;
    std::cout << "Enter the number of elements in the array [2-" << max_threads_num << "]: ";
    std::string input;
    std::cin >> N;

    if (N < 2 || N > max_threads_num){
      std::cout << "Invalid input." << std::endl;
      continue;
    }

    int real_size_power = log2_ceil(N);
    real_size = 1 << real_size_power;

    int* x = nullptr;
    hipMallocManaged(&x, real_size*sizeof(int));

    assign_random<<<1, N>>>(x, time(NULL));
    assign_constant_from<<<1, real_size-N>>>(x, std::numeric_limits<int>::max(), N);
    hipDeviceSynchronize();
    std::cout<<"Before sorting: ";
    print_array(x, N);

    bitonic_sort(x, real_size);
    std::cout<<"After sorting: ";
    print_array(x, N);
    hipFree(x);
  }
}